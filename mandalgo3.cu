#include "hip/hip_runtime.h"
#include "cuda_call.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include "fractol.h"
#include <stdio.h>
#ifdef __cplusplus
extern "C"
#endif

static void	handleerrorm(hipError_t err, const char *file, int line)
{
	if (err != hipSuccess)
	{
		exit(EXIT_FAILURE);
	}
}

__global__ void kernelm(double c0, double c1, double zoom, char *dev_img, double img_x, double img_y, int iter_max)
{
	double 	x = threadIdx.x+blockIdx.x*blockDim.x;
	double 	y = threadIdx.y+blockIdx.y*blockDim.y;
	double	zr;
	double	zi;
	int		i;
	double	tmp;
	int		pixel;
	int mapping[16][3];

	mapping[0][0] = 66;
	mapping[0][1] = 30;
	mapping[0][2] = 15;
	mapping[1][0] = 25;
	mapping[1][1] = 7;
	mapping[1][2] = 26;
	mapping[2][0] = 9;
	mapping[2][1] = 1;
	mapping[2][2] = 47;
    mapping[3][0] = 4;
	mapping[3][1] = 4;
	mapping[3][2] = 73;
	mapping[4][0] = 0;
	mapping[4][1] = 7;
	mapping[4][2] = 100;
	mapping[5][0] = 12;
	mapping[5][1] = 44;
	mapping[5][2] = 138;
 	mapping[6][0] = 24;
	mapping[6][1] = 82;
	mapping[6][2] = 177;
	mapping[7][0] = 57;
	mapping[7][1] = 125;
	mapping[7][2] = 209;
    mapping[8][0] = 134;
	mapping[8][1] = 181;
	mapping[8][2] = 229;
    mapping[9][0] = 211;
	mapping[9][1] = 234;
	mapping[9][2] = 248;
    mapping[10][0] = 241;
	mapping[10][1] = 233;
	mapping[10][2] = 191;
	mapping[11][0] = 248;
	mapping[11][1] = 201;
	mapping[11][2] = 95;
	mapping[12][0] = 255;
	mapping[12][1] = 170;
	mapping[12][2] = 0;
	mapping[13][0] = 204;
	mapping[13][1] = 128;
	mapping[13][2] = 0;
	mapping[14][0] = 153;
	mapping[14][1] = 87;
	mapping[14][2] = 0;
	mapping[15][0] = 106;
	mapping[15][1] = 52;
	mapping[15][2] = 3;
	if (x > LEN + 1)
		return ;
	if (y > LEN + 1)
		return ;
	pixel = x * 4 + (y * LEN * 4) * 2;
	zr = 0;
	zi = 0;
	i = 0;
	c0 = x / zoom + img_x;
	c1 = y / zoom + img_y;

	while (sqrtf(zr * zr + zi * zi) < 4 && i < iter_max)
	{
		tmp = zr;
		zr = zr * zr - zi * zi + c0;
		zi = 2 * zi * tmp + c1;
		i++;
		if (i == iter_max)
		{
			dev_img[pixel] = 0;
			dev_img[pixel + 1] = 0;
			dev_img[pixel + 2] = 0;
			dev_img[pixel + 3] = 0;
			break ;
		}
		else
		{
			dev_img[pixel] = mapping[i % 16][0];
			dev_img[pixel + 1] = mapping[i % 16][1];
			dev_img[pixel + 2] = mapping[i % 16][2];
			dev_img[pixel + 3] = 0;
		}
	}
}

#include <stdio.h>
#ifdef __cplusplus
extern "C"
#endif

char	*callcuda_mand(t_data *data, double x, double y)
{
	char 	*dev_img = NULL;
	double	c[2];
	double	zoo;
	double	img_x;
	double	img_y;

	img_x = data->x1;
	img_y = data->y1;
	c[0] = (x + (LEN / 2)) / (LEN / 2);
	c[1] = (y + (LEN / 2)) / (LEN / 2);

	HANDLE_ERRORM(hipMalloc((void**)&dev_img, (LEN * LEN * 4 * 2 + 1)));
	dim3	blocks((LEN)/ 32, (LEN) / 32);
	dim3	threads(32, 32);
	zoo = data->zoom;
	kernelm<<<blocks,threads>>>(c[0], c[1], zoo, dev_img, img_x, img_y, data->iter_max);
	hipDeviceSynchronize();
	HANDLE_ERRORM(hipMemcpy(data->img_str, dev_img, LEN * LEN * 4 * 2 + 1, hipMemcpyDeviceToHost));
	hipFree(dev_img);
	return (data->img_str);
}